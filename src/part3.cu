#include "hip/hip_runtime.h"
/*	
 * noise_remover.cpp
 *
 * This program removes noise from an image based on Speckle Reducing Anisotropic Diffusion
 * Y. Yu, S. Acton, Speckle reducing anisotropic diffusion, 
 * IEEE Transactions on Image Processing 11(11)(2002) 1260-1270 <http://people.virginia.edu/~sc5nf/01097762.pdf>
 * Original implementation is Modified by Burak BASTEM
 */

 #include <stdlib.h>
 #include <stdio.h>
 #include <math.h>
 #include <string.h>
 #include <sys/time.h>
 #define STB_IMAGE_IMPLEMENTATION
 #include "stb_image.h"
 #define STB_IMAGE_WRITE_IMPLEMENTATION
 #include "stb_image_write.h"
 
 #define MATCH(s) (!strcmp(argv[ac], (s)))
 
 // returns the current time
 static const double kMicro = 1.0e-6;
 double get_time() {
     struct timeval TV;
     struct timezone TZ;
     const int RC = gettimeofday(&TV, &TZ);
     if(RC == -1) {
         printf("ERROR: Bad call to gettimeofday\n");
         return(-1);
     }
     return( ((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec) );
 }

 __global__ void compute1(float* image, float* diff_coef, float std_dev, int width, int height,
                            float* north, float* south, float* east, float* west)
 {
    int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int index = row * width + col;

    if(row < height - 1 && col < width - 1){

        float image_k = image[index];

        float north_k = image[index - width] - image_k;
        float south_k = image[index + width] - image_k;
        float west_k = image[index - 1] - image_k;
        float east_k = image[index + 1] - image_k;

        float gradient_square = ( north_k * north_k 
                                + south_k * south_k
                                + west_k  * west_k 
                                + east_k  * east_k ) / (image_k * image_k);
        float laplacian = (north_k + south_k + west_k + east_k) / image_k;
        float num = (0.5 * gradient_square) - ((1.0 / 16.0) * (laplacian * laplacian));
        float den = 1 + (.25 * laplacian); 
        float std_dev2 = num / (den * den); 
        den = (std_dev2 - std_dev) / (std_dev * (1 + std_dev)); 
        float diff_coef_k = 1.0 / (1.0 + den);

        north[index] = north_k;
        south[index] = south_k;
        east[index]  = east_k;
        west[index]  = west_k;

        if (diff_coef_k < 0) {
            diff_coef[index] = 0;
        } else if (diff_coef_k > 1){
            diff_coef[index] = 1;
        } else {
            diff_coef[index] = diff_coef_k;
        }
    }

 }

 __global__ void compute2(float* image, float* diff_coef, float* north, float* south,
                                float* east, float* west, float lambda, int width, int height)
{
    __shared__ float temp[blockDim.y + 2][blockDim.x + 2];

    int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int g_index = row * width + col;
    int tx = threadIdx.x + 1, ty = threadIdx.y + 1;

    if(col - 1 >= width || row - 1 >= height) return;

    temp[ty][tx] = diff_coef[g_index];

    if(threadIdx.x < 1 || threadIdx.y < 1)
    {
        temp[ty - 1][tx - 1] = diff_coef[g_index - width - 1];
    }

    __syncthreads();

    float diff_coef_north = temp[ty][tx];	
    float diff_coef_south = temp[ty + 1][tx];	
    float diff_coef_west = temp[ty][tx];	        
    float diff_coef_east = temp[ty][tx + 1];					
    float divergence = diff_coef_north * north[g_index] 
                        + diff_coef_south * south[g_index] 
                        + diff_coef_west * west[g_index] 
                        + diff_coef_east * east[g_index];

    image[g_index] = image[g_index] + 0.25 * lambda * divergence;
    

}

// __global__ void reduction(float* image, float* sums, float* sums2, int size, int numblocks)
// {
//     __shared__ float sdata[numblocks];
//     __shared__ float sdata2[numblocks];

//     unsigned int tid = threadIdx.x;
//     unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    
//     float image_i = (i < size) ? image[i] : 0;
//     float mySum = image_i;
//     float mySum2 = image_i * image_i;

//     float image_j = (i + blockDim.x < size) ? image[i + blockDim.x] : 0;
//     mySum += image_j;
//     mySum2 += image_j * image_j;
    
//     sdata[tid] = mySum;
//     sdata2[tid] = mySum2;
//     __syncthreads();

//     for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
//         if (tid < s) {
//           sdata[tid] = mySum = mySum + sdata[tid + s];
//           sdata2[tid] = mySum2 = mySum2 + sdata2[tid + s];
//         }
//         __syncthreads();
//     }

//     if (tid == 0){
//         sums[blockIdx.x] = mySum;
//         sums2[blockIdx.x] = mySum2;
//     }
// }
 
 int main(int argc, char *argv[]) {
     // Part I: allocate and initialize variables
     double time_0, time_1, time_2, time_3, time_4, time_5, time_6, time_7, time_8;	// time variables
     time_0 = get_time();
     const char *filename = "input.pgm";
     const char *outputname = "output.png";	
     int width, height, pixelWidth, n_pixels;
     int n_iter = 50;
     float lambda = 0.5;
     float mean, variance, std_dev;	//local region statistics
     float *north_deriv, *south_deriv, *west_deriv, *east_deriv; // directional derivatives
     float *north_deriv_dev, *south_deriv_dev, *west_deriv_dev, *east_deriv_dev; // device derivatives
     float tmp, sum, sum2;	// calculation variables
     float gradient_square, laplacian, num, den, std_dev2, divergence;	// calculation variables
     float *diff_coef, *diff_coef_dev;	// diffusion coefficient
     float diff_coef_north, diff_coef_south, diff_coef_west, diff_coef_east;	// directional diffusion coefficients
     unsigned char *image_dev;
     long k;	// current pixel index
     time_1 = get_time();	
     
     // Part II: parse command line arguments
     if(argc<2) {
       printf("Usage: %s [-i < filename>] [-iter <n_iter>] [-l <lambda>] [-o <outputfilename>]\n",argv[0]);
       return(-1);
     }
     for(int ac=1;ac<argc;ac++) {
         if(MATCH("-i")) {
             filename = argv[++ac];
         } else if(MATCH("-iter")) {
             n_iter = atoi(argv[++ac]);
         } else if(MATCH("-l")) {
             lambda = atof(argv[++ac]);
         } else if(MATCH("-o")) {
             outputname = argv[++ac];
         } else {
         printf("Usage: %s [-i < filename>] [-iter <n_iter>] [-l <lambda>] [-o <outputfilename>]\n",argv[0]);
         return(-1);
         }
     }
     time_2 = get_time();
 
     // Part III: read image	
     printf("Reading image...\n");
     unsigned char *image = stbi_load(filename, &width, &height, &pixelWidth, 0);
     if (!image) {
         fprintf(stderr, "Couldn't load image.\n");
         return (-1);
     }
     printf("Image Read. Width : %d, Height : %d, nComp: %d\n",width,height,pixelWidth);
     n_pixels = height * width;
     time_3 = get_time();
 
     // Part IV: allocate variables
     
     hipMalloc((void**)&north_deriv_dev, sizeof(float) * n_pixels);
     hipMalloc((void**)&south_deriv_dev, sizeof(float) * n_pixels);
     hipMalloc((void**)&west_deriv_dev, sizeof(float) * n_pixels);
     hipMalloc((void**)&east_deriv_dev, sizeof(float) * n_pixels);
     hipMalloc((void**)&diff_coef_dev, sizeof(float) * n_pixels);
     hipMalloc((void**)&image_dev, sizeof(unsigned char) * n_pixels);

     hipMemcpy(image_dev, image, sizeof(unsigned char) * n_pixels, hipMemcpyHostToDevice);

     const int reduction_blocks = n_pixels/256 + (n_pixels % 256 == 0 ? 0 : 1);
     const int block_row = height/16 + (height % 256 == 0 ? 0 : 1);
     const int block_col = width/16 + (width % 256 == 0 ? 0 : 1);
     const dim3 blocks(block_row, block_col, 1), threads(16,16,1);

    //  float *sums, *sums2, *sums_dev, *sums_dev_2;
    //  sums = (float*) malloc(sizeof(float) * reduction_blocks);
    //  sums_2 = (float*) malloc(sizeof(float) * reduction_blocks);
    //  hipMalloc((void**)&sums_dev, sizeof(float)*reduction_blocks);
    //  hipMalloc((void**)&sums_dev_2, sizeof(float)*reduction_blocks);

     time_4 = get_time();
     // Part V: compute --- n_iter * (3 * height * width + 42 * (height-1) * (width-1) + 6) floating point arithmetic operations in totaL
     for (int iter = 0; iter < n_iter; iter++) {
         sum = 0;
         sum2 = 0;
         // REDUCTION AND STATISTICS
         // --- 3 floating point arithmetic operations per element -> 3*height*width in total
        //  reduction<<<reduction_blocks, 256>>>(image_dev, sums_dev, sums_dev_2, n_pixels reduction_blocks);

        //  hipMemcpy(sums, sums_dev, sizeof(float)*reduction_blocks, hipMemcpyDeviceToHost);
        //  hipMemcpy(sums2, sums_dev_2, sizeof(float)*reduction_blocks, hipMemcpyDeviceToHost);

        //  for(int i=0; i < reduction_blocks; i++){
        //     sum += sums[i];
        //     sum2 += sums2[i];
        //  }

         mean = sum / n_pixels; // --- 1 floating point arithmetic operations
         variance = (sum2 / n_pixels) - mean * mean; // --- 3 floating point arithmetic operations
         std_dev = variance / (mean * mean); // --- 2 floating point arithmetic operations
 
         //COMPUTE 1
         // --- 32 floating point arithmetic operations per element -> 32*(height-1)*(width-1) in total
         compute1<<<blocks, thread>>>(image_dev, diff_coef_dev, std_dev, width, height,
            north_deriv_dev, south_deriv_dev, east_deriv_dev, west_deriv_dev);

         // COMPUTE 2
         // divergence and image update --- 10 floating point arithmetic operations per element -> 10*(height-1)*(width-1) in total
         compute2<<<blocks, threads>>>(image_dev, diff_coef_dev, north_deriv_dev, south_deriv_dev,
            east_deriv_dev, west_deriv_dev, lambda, width, height);

     }
     hipMemcpy(image, image_dev, sizeof(unsigned char)*n_pixels, hipMemcpyDeviceToHost);
     time_5 = get_time();
 
     // Part VI: write image to file
     stbi_write_png(outputname, width, height, pixelWidth, image, 0);
     time_6 = get_time();
 
     // Part VII: get average of sum of pixels for testing and calculate GFLOPS
     // FOR VALIDATION - DO NOT PARALLELIZE
     float test = 0;
     for (int i = 0; i < height; i++) {
             for (int j = 0; j < width; j++) {
                 test += image[i * width + j];
         }
     }
     test /= n_pixels;	
 
     float gflops = (float) (n_iter * 1E-9 * (3 * height * width + 42 * (height-1) * (width-1) + 6)) / (time_5 - time_4);
     time_7 = get_time();
 
     // Part VII: deallocate variables
     stbi_image_free(image);
     hipFree(north_deriv_dev);
     hipFree(south_deriv_dev);
     hipFree(east_deriv_dev);
     hipFree(west_deriv_dev);
     hipFree(diff_coef_dev);
     hipFree(image_dev);
    //  hipFree(sums_dev);
    //  hipFree(sums_dev_2);
     time_8 = get_time();
 
     // print
     printf("Time spent in different stages of the application:\n");
     printf("%9.6f s => Part I: allocate and initialize variables\n", (time_1 - time_0));
     printf("%9.6f s => Part II: parse command line arguments\n", (time_2 - time_1));
     printf("%9.6f s => Part III: read image\n", (time_3 - time_2));
     printf("%9.6f s => Part IV: allocate variables\n", (time_4 - time_3));
     printf("%9.6f s => Part V: compute\n", (time_5 - time_4));
     printf("%9.6f s => Part VI: write image to file\n", (time_6 - time_5));
     printf("%9.6f s => Part VII: get average of sum of pixels for testing and calculate GFLOPS\n", (time_7 - time_6));
     printf("%9.6f s => Part VIII: deallocate variables\n", (time_7 - time_6));
     printf("Total time: %9.6f s\n", (time_8 - time_0));
     printf("Average of sum of pixels: %9.6f\n", test);
     printf("GFLOPS: %f\n", gflops);
     return 0;
 }
 
 